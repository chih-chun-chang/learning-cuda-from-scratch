
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#define BLOCK_SIZE 128

// Reduction 6: Complete Unrolling
// How can we unroll for block size that we don't know at compile time? => CUDA
// supports C++ template  parameters on device and host functions
template <unsigned int blockSize>
__device__ void warpReduce(volatile int* s_data, int tid) { 
  //volatile: tell the compiler not optimize
  if (blockSize >=64)
    s_data[tid] += s_data[tid + 32];
  if (blockSize >=32)
    s_data[tid] += s_data[tid + 16];
  if (blockSize >=16)
    s_data[tid] += s_data[tid + 8];
  if (blockSize >=8)
    s_data[tid] += s_data[tid + 4];
  if (blockSize >=4)
    s_data[tid] += s_data[tid + 2];
  if (blockSize >=2)
    s_data[tid] += s_data[tid + 1];
}

template <unsigned int blockSize>
__global__ void reduce5(int *g_idata, int *g_odata, size_t N) {
  extern __shared__ int s_data[]; //dynamically

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockDim.x*2) + threadIdx.x;
  s_data[tid] = 0;
  if ( i < N) 
    s_data[tid] = g_idata[i] + g_idata[i + blockDim.x];
  __syncthreads();

  if (blockSize >=512) {
    if (tid < 256) {
      s_data[tid] += s_data[tid + 256];
    }
    __syncthreads();
  }
  if (blockSize >=256) {
    if (tid < 128) {
      s_data[tid] += s_data[tid + 128];
    }
    __syncthreads();
  }
  if (blockSize >=128) {
    if (tid < 64) {
      s_data[tid] += s_data[tid + 64];
    }
    __syncthreads();
  }

  if (tid < 32)
    warpReduce<blockSize>(s_data, tid);
  if (tid == 0) 
    atomicAdd(&g_odata[0], s_data[0]);
}

void sum_by_cpu(int *input, int *output, size_t N) {
  int sum = 0;
  for (int i = 0; i < N; i++) {
    sum += input[i];
  }
  *output = sum;
}

int main(int argc, char* argv[]) {

  if (argc != 3) {
    std::cerr<< "usage: ./a.out N BLOCK_SIZE\n";
    std::exit(EXIT_FAILURE);
  }

  srand(time(NULL));
  size_t N = std::atoi(argv[1]);
  int *input = new int[N];
  int *output = new int[N];

  for (size_t i = 0; i < N; i++) {
    input[i] = rand() % 10;
  }

  // to store the result from host and device
  int cpu_sum, gpu_sum;

  // alocate memory in the device
  int *d_idata;
  int *d_odata;
  
  hipMalloc((void **) &d_idata, sizeof(int)*N);
  hipMalloc((void **) &d_odata, sizeof(int)*N);

  hipMemcpy(d_idata, input, sizeof(int)*N, hipMemcpyHostToDevice);

  // timing
  float elapsed_time_gpu, elapsed_time_cpu;
  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);

  // cpu
  hipEventRecord(beg, 0);
  sum_by_cpu(input, &cpu_sum, N);
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed_time_cpu, beg, end);
  printf("CPU time: %.3f ms \n", elapsed_time_cpu);

  // set grid size and block size
  //unsigned int block_sz = BLOCK_SIZE;
  //unsigned int grid_sz = (N + block_sz - 1) / block_sz;

  // gpu
  int threads = std::atoi(argv[2]);
  unsigned int dimBlock = threads;
  unsigned int dimGrid = (N + dimBlock - 1) / dimBlock;
  
  hipEventRecord(beg, 0);

  switch (threads) {
    case 512:
      reduce5<512><<< dimGrid, dimBlock >>>(d_idata, d_odata, N); break;
    case 256:
      reduce5<256><<< dimGrid, dimBlock >>>(d_idata, d_odata, N); break;
    case 128:
      reduce5<128><<< dimGrid, dimBlock >>>(d_idata, d_odata, N); break;
    case 64:
      reduce5< 64><<< dimGrid, dimBlock >>>(d_idata, d_odata, N); break;
    case 32:
      reduce5< 32><<< dimGrid, dimBlock >>>(d_idata, d_odata, N); break;
    case 16:
      reduce5< 16><<< dimGrid, dimBlock >>>(d_idata, d_odata, N); break;
    case 8:
      reduce5<  8><<< dimGrid, dimBlock >>>(d_idata, d_odata, N); break;
    case 4:
      reduce5<  4><<< dimGrid, dimBlock >>>(d_idata, d_odata, N); break;
    case 2:
      reduce5<  2><<< dimGrid, dimBlock >>>(d_idata, d_odata, N); break;
    case 1:
      reduce5<  1><<< dimGrid, dimBlock >>>(d_idata, d_odata, N); break;
    default: break;
  }

  hipMemcpy(&gpu_sum, d_odata, sizeof(int), hipMemcpyDeviceToHost);
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed_time_gpu, beg, end);
  printf("GPU time: %.3f ms \n", elapsed_time_gpu);

  printf("Reduction #5 -> Match: %s \n", cpu_sum==gpu_sum ? "True" : "False");
  printf("%d %d\n", cpu_sum, gpu_sum);

  return 0;  
  
}
