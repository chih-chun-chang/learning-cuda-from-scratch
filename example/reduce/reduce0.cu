
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#define BLOCK_SIZE 128

// Reduction #1: Interleaved Addressing
__global__ void reduce0(int *g_idata, int *g_odata) {
  extern __shared__ int s_data[]; //dynamically

  // each thread loads one element from global to shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  s_data[tid] = g_idata[i];
  
  __syncthreads();

  // do reduction in shared mem
  // 1st round: (stride = 1)
  //    tid = 0, 2, 4, 6, 8
  // 2nd round: (stride = 2)
  //    tid = 0, 4, 8
  // 3rd round: (stride = 4)
  //    tid = 0, 8
  // 4nd round: (stride = 8)
  //    tid = 0
  for (unsigned int s=1; s < blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      // ------------------------------------
      // Problem Here!!
      // * Highly divergent
      // => warps are very inefficient
      // * % operator is very slow
      // ------------------------------------
      s_data[tid] += s_data[tid+s];
    }
    __syncthreads();
  }

  // write the result for this block to global mem
  if (tid == 0)
    atomicAdd(&g_odata[0], s_data[0]);
    //g_odata[0] += s_data[0];
}


void sum_by_cpu(int *input, int *output, size_t N) {
  int sum = 0;
  for (int i = 0; i < N; i++) {
    sum += input[i];
  }
  *output = sum;
}

int main(int argc, char* argv[]) {

  if (argc != 2) {
    std::cerr<< "usage: ./a.out N\n";
    std::exit(EXIT_FAILURE);
  }

  srand(time(NULL));
  size_t N = std::atoi(argv[1]);
  int *input = new int[N];
  int *output = new int[N];

  for (size_t i = 0; i < N; i++) {
    input[i] = rand() % 10;
  }

  // to store the result from host and device
  int cpu_sum, gpu_sum;

  // alocate memory in the device
  int *d_input;
  int *d_output;
  
  hipMalloc((void **) &d_input, sizeof(int)*N);
  hipMalloc((void **) &d_output, sizeof(int)*N);

  hipMemcpy(d_input, input, sizeof(int)*N, hipMemcpyHostToDevice);

  // timing
  float elapsed_time_gpu, elapsed_time_cpu;
  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);

  // cpu
  hipEventRecord(beg, 0);
  sum_by_cpu(input, &cpu_sum, N);
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed_time_cpu, beg, end);
  printf("CPU time: %.3f ms \n", elapsed_time_cpu);

  // set grid size and block size
  unsigned int block_sz = BLOCK_SIZE;
  unsigned int grid_sz = (N + block_sz - 1) / block_sz;

  // gpu
  hipEventRecord(beg, 0);
  reduce0<<<grid_sz, block_sz>>>(d_input, d_output);
  hipMemcpy(&gpu_sum, d_output, sizeof(int), hipMemcpyDeviceToHost);
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed_time_gpu, beg, end);

  printf("GPU time: %.3f ms \n", elapsed_time_gpu); 

  printf("Reduction #0() Match: %s \n", cpu_sum==gpu_sum ? "True" : "False");

  return 0;  
  
}
